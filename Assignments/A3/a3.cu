#include "hip/hip_runtime.h"
#include <stdio.h>
#include "file1.hpp"

__global__ void mykernel(){
  printf("Hello from mykernel\n");
}

void hello(){
  mykernel<<<1,1>>>();
  hipDeviceSynchronize();
}
